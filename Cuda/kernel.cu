#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <omp.h>

#include <stdio.h>
#include <string>
#include <vector>
#include <filesystem>

#include "logging.h"

#include "imageIO.h"
#include "cudaCrop.h"
#include "cudaMappedMemory.h"

namespace fs = std::filesystem;

class CudaMemory {
public:
	CudaMemory(uchar3* ptr) : ptr_(ptr) {}
	~CudaMemory() { hipFree(ptr_); }
private:
	uchar3* ptr_;
};

static std::vector<fs::path> listPngPaths(const std::string& directory)
{
	std::vector<fs::path> paths;
	const fs::path p(directory);
	for (const fs::path p : fs::directory_iterator(p)) {
		if (fs::is_regular_file(p) && p.extension() == ".png") {
			paths.push_back(p);
		}
	}
	return paths;
}

void processImage(const char* input_path, const char* output_path)
{
	uchar3* input_image = NULL;
	uchar3* output_image = NULL;
	CudaMemory input_memory(input_image), output_memory(output_image);

	int width = 0, height = 0;

	// Load image
	if (!loadImage(input_path, &input_image, &width, &height)) {
		fprintf(stderr, "[%s] loadImage failed!", input_path);
		return;
	}

	int crop_width = width / 2;
	int crop_height = height / 3;

	// Allocate output image
	if (!cudaAllocMapped(&output_image, sizeof(uchar3) * crop_width * crop_height)) {
		fprintf(stderr, "[%s] cudaAllocMapped failed!", input_path);
		return;
	}

	// Crop image
	int4 roi = {
		width / 4,
		height / 3,
		width / 4 + crop_width,
		height / 3 + crop_height
	};
	hipError_t cudaStatus = cudaCrop(
		input_image,
		output_image,
		roi,
		width,
		height
	);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "[%s] cudaCrop failed! cudaStatus=%d", input_path, cudaStatus);
		return;
	}

	// Save image
	if (!saveImage(output_path, output_image, crop_width, crop_height)) {
		fprintf(stderr, "[%s] saveImage failed!", input_path);
		return;
	}
}

int main()
{
	Log::SetLevel(Log::Level::SILENT);

	const fs::path input_directory = R"(C:\Users\owner\source\repos\CropBenchmark\images)";

	// input_directory\cropped ディレクトリを初期化
	// 存在していたら削除して再作成、存在していなかったら作成
	const fs::path output_directory = input_directory / "cropped";
	if (fs::exists(output_directory)) {
		fs::remove_all(output_directory);
	}
	fs::create_directory(output_directory);

	const std::vector<fs::path> images = listPngPaths(input_directory.string());
	const int size = images.size();

	// ここから時間計測
	const clock_t start = clock();

#pragma omp parallel for
	for (int i = 0; i < size; i++) {
		const fs::path input_path = images[i];
		//printf("[%d]%s\n", omp_get_thread_num(), input_path.string().c_str());
		const fs::path output_path = output_directory / input_path.filename();
		processImage(input_path.string().c_str(), output_path.string().c_str());
	}

	const clock_t end = clock();
	const double time = (double)(end - start) / CLOCKS_PER_SEC;
	printf("処理時間   : %f\n", time);
	printf("[files/s]  : %f\n", size / time);
	printf("[s/files]  : %f\n", time / size);

	return 0;
}
